#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void monte_carlo_kernel(float* results, int iterations, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < iterations) {
        hiprandState state;
        hiprand_init(seed + idx, 0, 0, &state);
        
        float x = hiprand_uniform(&state);
        float y = hiprand_uniform(&state);
        
        results[idx] = (x*x + y*y <= 1.0f) ? 1.0f : 0.0f;
    }
}

extern "C" float estimate_pi(int iterations) {
    float *d_results, *h_results;
    h_results = (float*)malloc(iterations * sizeof(float));
    hipMalloc(&d_results, iterations * sizeof(float));

    int threadsPerBlock = 256;
    int blocks = (iterations + threadsPerBlock - 1) / threadsPerBlock;

    monte_carlo_kernel<<<blocks, threadsPerBlock>>>(d_results, iterations, time(NULL));

    hipMemcpy(h_results, d_results, iterations * sizeof(float), hipMemcpyDeviceToHost);

    float sum = 0.0f;
    for (int i = 0; i < iterations; i++) {
        sum += h_results[i];
    }

    float pi = 4.0f * sum / iterations;

    free(h_results);
    hipFree(d_results);

    return pi;
}